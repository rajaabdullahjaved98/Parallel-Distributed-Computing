#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include "Timer.h"

// CUDA kernel for matrix multiplication
__global__ void matrixMultiplication(int *A, int *B, int *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {

    Timer gpuMulTime, cpuTime;
    initTimer(&gpuMulTime, "GPU Multiplication Time");
    initTimer(&cpuTime, "CPU Time for Initialization and Allocation: ");
    int N = 1024; // Size of the matrices

    int *h_A, *h_B, *h_C; // Host matrices
    int *d_A, *d_B, *d_C; // Device matrices

    //OMP Section for Host Memory Allocation and Data Initialization
    startTimer(&cpuTime);
    #pragma omp parallel sections
    {
    	#pragma omp section
    	{	
    		// Allocate host memory
    		h_A = (int*)malloc(N * N * sizeof(int));
    		h_B = (int*)malloc(N * N * sizeof(int));
    		h_C = (int*)malloc(N * N * sizeof(int));
    	}
    	
    	#pragma omp section
    	{
    		// Initialize host matrices with some values
    		for (int i = 0; i < N * N; i++) 
    		{
        		h_A[i] = i;
        		h_B[i] = i;
    		}		
    	}
    }	
    stopTimer(&cpuTime);

    // Allocate device memory
    hipMalloc((void**)&d_A, N * N * sizeof(int));
    hipMalloc((void**)&d_B, N * N * sizeof(int));
    hipMalloc((void**)&d_C, N * N * sizeof(int));

    // Copy host matrices to device
    hipMemcpy(d_A, h_A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    startTimer(&gpuMulTime);
    matrixMultiplication<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    stopTimer(&gpuMulTime);

    // Copy result back to host
    hipMemcpy(h_C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print result (optional)
    /*for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", h_C[i * N + j]);
        }
        printf("\n");
    }*/

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printTimer(gpuMulTime);
    printTimer(cpuTime);

    return 0;
}